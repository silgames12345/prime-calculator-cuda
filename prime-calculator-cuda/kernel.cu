#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""
#include "kernel.cuh"

#include <iostream>


__global__ void findPrimes(int* primesArr, int* numbers, int* primesAnwser, int* primesSize) {
	int j = threadIdx.x;
	bool isDivideble = false;
	for (int i = 0; i < primesSize[0]; i++) {
		if (numbers[j] % primesArr[i] == 0) {
			isDivideble = true;
		}
	}
	if (!isDivideble) {
		primesAnwser[j] = numbers[j];
	}
}

std::vector<int> cudaPrimes(int* numbers, int numbersSize, std::vector<int> primes) {
	//converting vector to array
	int* primesArr = new int[primes.size()];
	std::copy(primes.begin(), primes.end(), primesArr);

	int primesArrSize = primes.size();
	int* primesArrSizePointer = &primesArrSize;

	int primesAnwser[THREADS] = {0};

	int* cudaNumbers = 0;
	int* cudaPrimes = 0;
	int* cudaPrimesAnwser = 0;

	int* cudaPrimesSize = 0;

	hipMalloc(&cudaNumbers, numbersSize);
	hipMalloc(&cudaPrimes, primesArrSize * sizeof(int));
	hipMalloc(&cudaPrimesAnwser, sizeof(primesAnwser));
	hipMalloc(&cudaPrimesSize, sizeof(primesArrSize));

	hipMemcpy(cudaNumbers, numbers, numbersSize, hipMemcpyHostToDevice);
	hipMemcpy(cudaPrimes, primesArr, primesArrSize*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(cudaPrimesAnwser, primesAnwser, sizeof(primesAnwser), hipMemcpyHostToDevice);
	hipMemcpy(cudaPrimesSize, primesArrSizePointer, sizeof(primesArrSize), hipMemcpyHostToDevice);

	findPrimes << <1, THREADS >> > (cudaPrimes, cudaNumbers, cudaPrimesAnwser, cudaPrimesSize);

	hipMemcpy(primesAnwser, cudaPrimesAnwser, sizeof(primesAnwser), hipMemcpyDeviceToHost);

	for (int i = 0; i < THREADS; i++) {
		if (primesAnwser[i] != 0) {
			primes.push_back(primesAnwser[i]);
		}
	}

	delete[] primesArr;

	return primes;
}

